#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "dnn.hpp"

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  #define Tn 16
  #define Ti 16
#endif

#define BATCH 16

VTYPE (*synapse)[BATCH][Nn][Ni];
VTYPE (*neuron_i)[BATCH][Ni];
VTYPE (*neuron_n)[BATCH][Nn];
VTYPE (*neuron_n2)[BATCH][Nn];
VTYPE (*neuron_n3)[BATCH][Nn];

void classifier(const VTYPE synapse[Nn][Ni],
                const VTYPE neuron_i[Ni],
                VTYPE neuron_n[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp = 0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}

void classifier_tiled(const VTYPE synapse[Nn][Ni],
                      const VTYPE neuron_i[Ni],
                      VTYPE neuron_n[Nn]) {
  VTYPE sum[Nn] = {};
  for (int outer_n = 0; outer_n < Nn; outer_n += Tnn) { // tiling for output neurons;
    for (int outer_i = 0; outer_i < Ni; outer_i += Tii) { // tiling for input neurons;
      for (int inner_n = outer_n; inner_n < outer_n + Tnn; inner_n += Tn) {
        for (int inner_i = outer_i; inner_i < outer_i + Tii; inner_i += Ti) {
          // Original code
          for (int n = inner_n; n < inner_n + Tn; n++) {
            VTYPE sum_sc = 0;
            for (int i = inner_i; i < inner_i + Ti; i++) {
              sum_sc += synapse[n][i] * neuron_i[i];
            }
            sum[n] += sum_sc;
          }
        }
      }
    }
    for (int n = outer_n; n < outer_n + Tnn; n++) {
      neuron_n[n] = transfer(sum[n]);
    }
  }
}

__global__ void GPU_classifier(const VTYPE synapse[Nn][Ni],
                               const VTYPE neuron_i[Ni],
                               VTYPE neuron_n[Nn],
                               size_t pitch) {
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + n * pitch);
  if (n < Nn) {
    VTYPE sum = 0;
    for (int i = 0; i < Ni; i++) {
        sum += synapse_row[i] * neuron_i[i];
    }
    neuron_n[n] = GPU_transfer(sum);
  }
}

__global__ void GPU_classifier_tiled(const VTYPE synapse[Nn][Ni],
                                     const VTYPE neuron_i[Ni],
                                     VTYPE neuron_n[Nn],
                                     size_t pitch,
                                     int tiling_size) {
  VTYPE sum = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + row * pitch);
  for (int t = 0; t < Ni; t += tiling_size) {
    for (int i = t; i < t + tiling_size; ++i) {
      sum += synapse_row[i] * neuron_i[i];
    }
  }
   neuron_n[row] = GPU_transfer(sum);
}

__global__ void GPU_classifier_tiled_smem(const VTYPE synapse[Nn][Ni],
                                          const VTYPE neuron_i[Ni],
                                          VTYPE neuron_n[Nn],
                                          size_t pitch,
                                          int tiling_size) {
  extern __shared__ VTYPE p[];
  VTYPE *local_synapse = p;
  VTYPE *local_neuron_i = p + Tn * tiling_size;
  VTYPE sum = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + row * pitch);
  for (int t = 0; t < Ni; t += tiling_size) {
    int col = t * blockDim.x + threadIdx.x;

    local_synapse[threadIdx.y * tiling_size + threadIdx.x] = synapse_row[col];
    local_neuron_i[threadIdx.x] = neuron_i[col];

    __syncthreads();
    for (int i = t; i < t + tiling_size; ++i) {
      sum += local_synapse[threadIdx.y * tiling_size + i] * local_neuron_i[i];
    }
   __syncthreads();
  }
   neuron_n[row] = GPU_transfer(sum);
}

int main(void) {
  std::cout << "------ Initializing ------" << std::endl;

  synapse   = (VTYPE (*)[BATCH][Nn][Ni]) aligned_alloc(64, BATCH * Nn * Ni * sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[BATCH][Ni]) aligned_alloc(64, BATCH * Ni * sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[BATCH][Nn]) aligned_alloc(64, BATCH * Nn * sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[BATCH][Nn]) aligned_alloc(64, BATCH * Nn * sizeof(VTYPE));
  neuron_n3 = (VTYPE (*)[BATCH][Nn]) aligned_alloc(64, BATCH * Nn * sizeof(VTYPE));

  fill_random((VTYPE *) synapse, Nn * Ni * BATCH);
  fill_random((VTYPE *) neuron_i, Ni * BATCH);
  memset(neuron_n, 0, Nn * BATCH* sizeof(VTYPE));
  memset(neuron_n2, 0, Nn * BATCH * sizeof(VTYPE));
  memset(neuron_n3, 0, Nn * BATCH * sizeof(VTYPE));

  std::cout << "------ Running CPU version ------" << std::endl;
  std::cout << "Simple version: \t";

  timeit([]() {
    for(int b = 0; b < BATCH; ++b) {
      classifier((*synapse)[b], (*neuron_i)[b], (*neuron_n)[b]);
    }
  });

  std::cout << "Tiled version:  \t";  
  timeit([]() {
    for(int b = 0; b < BATCH; ++b) {
      classifier_tiled((*synapse)[b], (*neuron_i)[b], (*neuron_n2)[b]);
    }
  });

  compare((VTYPE *)neuron_n, (VTYPE *)neuron_n2, Nn * BATCH);

  std::cout << "------ Running GPU version ------" << std::endl;
  VTYPE (*d_synapse)[Ni];
  VTYPE *d_neuron_i, *d_neuron_n;

  size_t pitch;
  hipMallocPitch((void **)&d_synapse, &pitch, Ni * sizeof(VTYPE), Nn);
  hipMalloc(&d_neuron_i, Ni * sizeof(VTYPE));
  hipMalloc(&d_neuron_n, Nn * sizeof(VTYPE));

  hipMemcpy2D(d_synapse, pitch, synapse, Ni * sizeof(VTYPE), Ni * sizeof(VTYPE), Nn, hipMemcpyHostToDevice);
  hipMemcpy(d_neuron_i, neuron_i, Ni * sizeof(VTYPE), hipMemcpyHostToDevice);

  std::cout << "Simple version: \t";


  int num_threads = 8;
  int num_blocks = Nn / num_threads;

  hipMemset(d_neuron_n, 0, Nn * sizeof(VTYPE));
  CUDA_timeit([&]() {
    GPU_classifier<<<num_blocks, num_threads>>>(d_synapse, d_neuron_i, d_neuron_n, pitch);
  });
  hipMemcpy(neuron_n3, d_neuron_n, Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);

  compare(neuron_n, neuron_n3, Nn);
  std::cout << "Tiled version:\t";

/*  for(int ti = 2; ti <= 512; ti *= 2) {
  hipMemset(d_neuron_n, 0, Nn * sizeof(VTYPE));
  dim3 blockDim(1, Tn), gridDim(1, Nn/Tn);
  CUDA_timeit([&]() {
    GPU_classifier_tiled<<<gridDim, blockDim>>>(d_synapse, d_neuron_i, d_neuron_n, pitch, ti);
  });
  auto err = hipGetLastError();
  if(err) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  hipMemcpy(neuron_n2, d_neuron_n, Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);
  std::cout << "tiled size = " << ti << std::endl;

  compare(neuron_n, neuron_n2, Nn);
  }*/
}
