#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include "dnn.hpp"

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  //#define Tn 5
  //#define Ti 25
  #define Tn 16
  #define Ti 16
#endif

VTYPE synapse[Nn][Ni] __attribute__((aligned(64)));
VTYPE neuron_i[Ni] __attribute__((aligned(64)));
VTYPE neuron_n[Nn] __attribute__((aligned(64)));
VTYPE neuron_n2[Nn] __attribute__((aligned(64)));

void classifier_layer(VTYPE synapse[Nn][Ni],
                      VTYPE neuron_i[Ni],
                      VTYPE neuron_n[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp = 0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}


__global__ void GPU_classifier_layer(VTYPE synapse[Nn][Ni],
                                     VTYPE neuron_i[Ni],
                                     VTYPE neuron_n[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp = 0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = GPU_transfer(temp);
  }
}

void classifier_layer_blocked(VTYPE synapse[Nn][Ni],
                              VTYPE neuron_i[Ni], 
                              VTYPE neuron_n[Nn]) {
  VTYPE sum[Nn] = {};
  for (int outer_n = 0; outer_n < Nn; outer_n += Tnn) { // tiling for output neurons;
    for (int outer_i = 0; outer_i < Ni; outer_i += Tii) { // tiling for input neurons;
      for (int inner_n = outer_n; inner_n < outer_n + Tnn; inner_n += Tn) {
        for (int inner_i = outer_i; inner_i < iii + Tii; inner_i += Ti) {
          // Original code
          for (int n = inner_n; n < inner_n + Tn; n++) {
            VTYPE sum_sc = 0;
            for (int i = inner_i; i < inner_i + Ti; i++) {
              sum_sc += synapse[n][i] * neuron_i[i];
            }
            sum[n] += sum_sc;
          }
        }
      }
    }
    for (int n = outer_n; n < outer_n + Tnn; n++) {
      neuron_n[n] = transfer(sum[n]);
    }
  }
}

int main(void) {
  std::cout << "------ Running CPU version ------" << std::endl;

  fill_random((VTYPE *) synapse, Nn * Ni);
  fill_random((VTYPE *) neuron_i, Ni);
  memset(neuron_n, 0, Nn * sizeof(VTYPE));
  memset(neuron_n2, 0, Nn * sizeof(VTYPE));

  std::cout << "Simple version: \t";
  auto f1 = std::bind(classifier_layer, synapse, neuron_i, neuron_n);
  timeit(f1);

  std::cout << "Blocked version:\t";  
  auto f2 = std::bind(classifier_layer_blocked, synapse, neuron_i, neuron_n2);
  timeit(f2);

  compare(neuron_n, neuron_n2, Nn);

  std::cout << "------ Running GPU version ------" << std::endl;

  /* TODO: GPU version*/
}

