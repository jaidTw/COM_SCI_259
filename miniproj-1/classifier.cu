#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "dnn.hpp"

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  #define Tn 16
  #define Ti 16
#endif

VTYPE (*synapse)[Nn][Ni];
VTYPE (*neuron_i)[Ni];
VTYPE (*neuron_n)[Nn];
VTYPE (*neuron_n2)[Nn];
VTYPE (*neuron_n3)[Nn];

void classifier(const VTYPE synapse[Nn][Ni],
                const VTYPE neuron_i[Ni],
                VTYPE neuron_n[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp = 0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}

void classifier_tiled(const VTYPE synapse[Nn][Ni],
                      const VTYPE neuron_i[Ni],
                      VTYPE neuron_n[Nn]) {
  VTYPE sum[Nn] = {};
  for (int outer_n = 0; outer_n < Nn; outer_n += Tnn) { // tiling for output neurons;
    for (int outer_i = 0; outer_i < Ni; outer_i += Tii) { // tiling for input neurons;
      for (int inner_n = outer_n; inner_n < outer_n + Tnn; inner_n += Tn) {
        for (int inner_i = outer_i; inner_i < outer_i + Tii; inner_i += Ti) {
          // Original code
          for (int n = inner_n; n < inner_n + Tn; n++) {
            VTYPE sum_sc = 0;
            for (int i = inner_i; i < inner_i + Ti; i++) {
              sum_sc += synapse[n][i] * neuron_i[i];
            }
            sum[n] += sum_sc;
          }
        }
      }
    }
    for (int n = outer_n; n < outer_n + Tnn; n++) {
      neuron_n[n] = transfer(sum[n]);
    }
  }
}


__global__ void GPU_classifier(const hipPitchedPtr synapse,
                               const hipPitchedPtr neuron_i,
                               const hipPitchedPtr neuron_n,
                               size_t batch_begin,
                               int tn) {
  /* x => n, y => batch */
  int b = blockIdx.y + batch_begin;
  int n_begin = (blockIdx.x * blockDim.x + threadIdx.x) * tn;

  VTYPE * const neuron_i_row = (VTYPE *)((char *)neuron_i.ptr + b * neuron_i.pitch);
  VTYPE * const neuron_n_row = (VTYPE *)((char *)neuron_n.ptr + b * neuron_n.pitch);
  for (int n = n_begin; n < n_begin + tn; n++) {
    VTYPE sum = 0;
    VTYPE * const synapse_row = (VTYPE *)((char *)synapse.ptr + (b * Nn + n) * synapse.pitch);
    for (int i = 0; i < Ni; i++) {
        sum += synapse_row[i] * neuron_i_row[i];
    }
    neuron_n_row[n] = GPU_transfer(sum);
  }
}

__global__ void GPU_classifier_tiled(const VTYPE (*synapse)[Nn][Ni],
                                     const VTYPE (*neuron_i)[Ni],
                                     VTYPE (*neuron_n)[Nn],
                                     size_t pitch,
                                     size_t batch_begin,
                                     int ti) {
  uint32_t pitch_synapse = (pitch >> 40) & 0x0FFFFF;
  uint32_t pitch_neuron_i = (pitch >> 20) & 0x0FFFFF;
  uint32_t pitch_neuron_n = pitch & 0x0FFFFF;
  /* x => n, y => batch */
  int b = blockIdx.y + batch_begin;
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  VTYPE *neuron_i_row = (VTYPE *)((char *)neuron_i + b * pitch_neuron_i);
  VTYPE *neuron_n_row = (VTYPE *)((char *)neuron_n + b * pitch_neuron_n);
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + (b * Nn + n) * pitch_synapse);
  VTYPE sum = 0;
  for (int out_i = 0; out_i < Ni; out_i += ti) {
    for(int i = out_i; i < out_i + ti; i++) {
      sum += synapse_row[i] * neuron_i_row[i];
    }
  }
  neuron_n_row[n] = GPU_transfer(sum);
}

__global__ void GPU_classifier_tiled_smem(const VTYPE synapse[Nn][Ni],
                                          const VTYPE neuron_i[Ni],
                                          VTYPE neuron_n[Nn],
                                          size_t pitch,
                                          int tiling_size) {
  extern __shared__ VTYPE p[];
  VTYPE *local_synapse = p;
  VTYPE *local_neuron_i = p + Tn * tiling_size;
  VTYPE sum = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + row * pitch);
  for (int t = 0; t < Ni; t += tiling_size) {
    int col = t * blockDim.x + threadIdx.x;

    local_synapse[threadIdx.y * tiling_size + threadIdx.x] = synapse_row[col];
    local_neuron_i[threadIdx.x] = neuron_i[col];

    __syncthreads();
    for (int i = t; i < t + tiling_size; ++i) {
      sum += local_synapse[threadIdx.y * tiling_size + i] * local_neuron_i[i];
    }
   __syncthreads();
  }
   neuron_n[row] = GPU_transfer(sum);
}

int main(int argc, char **argv) {
  if (argc < 3) {
    std::cerr << "Usage : " << argv[0] << " BATCH_SIZE BATCH_IN_PARALLEL" << std::endl;
    exit(0);
  }
  const int batch = strtol(argv[1], nullptr, 10);
  const int batch_in_parallel = strtol(argv[2], nullptr, 10);
  if (batch_in_parallel > batch) {
    std::cerr << "BATCH_IN_PARALLEL must smaller than BATCH" << std::endl;
    exit(0);
  } else if (batch % batch_in_parallel) {
    std::cerr << "BATCH must be a multiple of BATCH_IN_PARALLEL" << std::endl;
    exit(0);
  }

  std::cout << "Initializing ..." << std::endl;

  synapse   = (VTYPE (*)[Nn][Ni]) aligned_alloc(64, batch * Nn * Ni * sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[Ni]) aligned_alloc(64, batch * Ni * sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[Nn]) aligned_alloc(64, batch * Nn * sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[Nn]) aligned_alloc(64, batch * Nn * sizeof(VTYPE));
  neuron_n3 = (VTYPE (*)[Nn]) aligned_alloc(64, batch * Nn * sizeof(VTYPE));

  fill_random((VTYPE *) synapse, Nn * Ni * batch);
  fill_random((VTYPE *) neuron_i, Ni * batch);
  memset(neuron_n, 0, Nn * batch * sizeof(VTYPE));
  memset(neuron_n2, 0, Nn * batch * sizeof(VTYPE));
  memset(neuron_n3, 0, Nn * batch * sizeof(VTYPE));

  std::cout << "CPU Simple version: \t\t\t\t\t";
  timeit([&]() {
    for(int b = 0; b < batch; ++b) {
      classifier(synapse[b], neuron_i[b], neuron_n[b]);
    }
  });

  std::cout << "CPU Tiled version:  \t\t\t\t\t";
  timeit([&]() {
    for(int b = 0; b < batch; ++b) {
      classifier_tiled(synapse[b], neuron_i[b], neuron_n2[b]);
    }
  });

  compare((VTYPE *)neuron_n, (VTYPE *)neuron_n2, Nn * batch);

  hipExtent extent_synapse = make_hipExtent(Ni * sizeof(VTYPE), Nn, batch);
  hipPitchedPtr d_synapse;
  MallocAndCpy3D(d_synapse, synapse, extent_synapse);

  size_t pitch_neuron_i, pitch_neuron_n;
  VTYPE (*_d_neuron_i)[Ni], (*_d_neuron_n)[Nn];
  hipMallocPitch((void **)&_d_neuron_i, &pitch_neuron_i, Ni * sizeof(VTYPE), batch);
  hipMallocPitch((void **)&_d_neuron_n, &pitch_neuron_n, Nn * sizeof(VTYPE), batch);
  hipPitchedPtr d_neuron_i = make_hipPitchedPtr(_d_neuron_i, pitch_neuron_i, Ni, batch);
  hipPitchedPtr d_neuron_n = make_hipPitchedPtr(_d_neuron_n, pitch_neuron_n, Nn, batch);
  hipMemcpy2D(d_neuron_i.ptr, d_neuron_i.pitch, neuron_i, Ni * sizeof(VTYPE), Ni * sizeof(VTYPE), batch, hipMemcpyHostToDevice);

  std::cout << "GPU version:\n";

  for(int tn = 1; tn < Nn; tn *= 2) {
    int threads_total = Nn / tn;
    for(int block_num = 1; block_num < threads_total; block_num *= 2) {
      dim3 block_size(threads_total / block_num);
      dim3 grid_size(block_num, batch_in_parallel);
      if (threads_total / block_num > 1024 || threads_total/block_num == 0)
        continue;

      hipMemset(d_neuron_n.ptr, 0, batch * Nn * sizeof(VTYPE));

      printf("Grid: (%4d, %4d), Block: (%4d, %4d), Tn=%4d\t", grid_size.x, grid_size.y, block_size.x, block_size.y, tn);
      CUDA_timeit([&]() {
        for(int b = 0; b < batch; b += batch_in_parallel) {
          GPU_classifier<<<grid_size, block_size>>>(d_synapse, d_neuron_i, d_neuron_n, b, tn);
        }
      });
      hipMemcpy2D(neuron_n3, Nn * sizeof(VTYPE), d_neuron_n.ptr, d_neuron_n.pitch, Nn * sizeof(VTYPE), batch, hipMemcpyDeviceToHost);

      cuda_check_error();
      compare((VTYPE *)neuron_n, (VTYPE *)neuron_n3, Nn * batch);
    }
  }
}
