#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "dnn.hpp"

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 128  // Number of Output Layers
  #define Ni 224  // Number of Input  Layers
#endif

#ifndef Tii
  // Tiling Sizes
  #define Tnn 32  
  #define Tii 32
  #define Tn 16
  #define Ti 16
#endif

VTYPE synapse[Nn][Ni] __attribute__((aligned(64)));
VTYPE neuron_i[Ni] __attribute__((aligned(64)));
VTYPE neuron_n[Nn] __attribute__((aligned(64)));
VTYPE neuron_n2[Nn] __attribute__((aligned(64)));
VTYPE neuron_n3[Nn] __attribute__((aligned(64)));

void classifier(const VTYPE synapse[Nn][Ni],
                const VTYPE neuron_i[Ni],
                VTYPE neuron_n[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE temp = 0;
    for (int i = 0; i < Ni; i++) {
      temp += synapse[n][i] * neuron_i[i];
    }
    neuron_n[n] = transfer(temp);
  }
}

void classifier_tiled(const VTYPE synapse[Nn][Ni],
                      const VTYPE neuron_i[Ni],
                      VTYPE neuron_n[Nn]) {
  VTYPE sum[Nn] = {};
  for (int outer_n = 0; outer_n < Nn; outer_n += Tnn) { // tiling for output neurons;
    for (int outer_i = 0; outer_i < Ni; outer_i += Tii) { // tiling for input neurons;
      for (int inner_n = outer_n; inner_n < outer_n + Tnn; inner_n += Tn) {
        for (int inner_i = outer_i; inner_i < outer_i + Tii; inner_i += Ti) {
          // Original code
          for (int n = inner_n; n < inner_n + Tn; n++) {
            VTYPE sum_sc = 0;
            for (int i = inner_i; i < inner_i + Ti; i++) {
              sum_sc += synapse[n][i] * neuron_i[i];
            }
            sum[n] += sum_sc;
          }
        }
      }
    }
    for (int n = outer_n; n < outer_n + Tnn; n++) {
      neuron_n[n] = transfer(sum[n]);
    }
  }
}

__global__ void GPU_classifier(const VTYPE synapse[Nn][Ni],
                               const VTYPE *neuron_i,
                               VTYPE *neuron_n,
                               size_t pitch) {
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + n * pitch);
  if (n < Nn) {
    VTYPE sum = 0;
    for (int i = 0; i < Ni; i++) {
        sum += synapse_row[i] * neuron_i[i];
    }
    neuron_n[n] = GPU_transfer(sum);
  }
}

__global__ void GPU_classifier_tiled(const VTYPE synapse[Nn][Ni],
                                      const VTYPE *neuron_i,
                                      VTYPE *neuron_n,
                                      size_t pitch) {
//  __shared__ float local_synapse[blockDim.y][blockDim.x];
//  __shared__ float local_neuron_i[blockDim.x];

  VTYPE sum = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  for (int t = 0; t < (Ni / blockDim.x); ++t) {
//    int col = t * blockDim.x + threadIdx.x;

//    local_synapse[threadIdx.y][threadIdx.x] = synapse[row][col];
//    local_neuron_i[threadIdx.x][threadIdx.y] = neuron_i;

//    __syncthreads();
    VTYPE *synapse_row = (VTYPE *)((char *)synapse + row * pitch);
    for (int i = 0; i < blockDim.x; ++i) {
      sum += synapse_row[i] * neuron_i[i];
    }
//   __syncthreads();
  }
   neuron_n[row] = GPU_transfer(sum);
}

int main(void) {
  std::cout << "------ Initializing ------" << std::endl;

  fill_random((VTYPE *) synapse, Nn * Ni);
  fill_random((VTYPE *) neuron_i, Ni);
  memset(neuron_n, 0, Nn * sizeof(VTYPE));
  memset(neuron_n2, 0, Nn * sizeof(VTYPE));
  memset(neuron_n3, 0, Nn * sizeof(VTYPE));

  std::cout << "------ Running CPU version ------" << std::endl;
  std::cout << "Simple version: \t";
  auto f1 = std::bind(classifier, synapse, neuron_i, neuron_n);
  timeit(f1);

  std::cout << "Blocked version:\t";  
  auto f2 = std::bind(classifier_tiled, synapse, neuron_i, neuron_n2);
  timeit(f2);

  compare(neuron_n, neuron_n2, Nn);

  std::cout << "------ Running GPU version ------" << std::endl;
  VTYPE (*d_synapse)[Ni];
  VTYPE *d_neuron_i, *d_neuron_n;

  size_t pitch;
  hipMallocPitch((void **)&d_synapse, &pitch, Ni * sizeof(VTYPE), Nn);
  hipMalloc(&d_neuron_i, Ni * sizeof(VTYPE));
  hipMalloc(&d_neuron_n, Nn * sizeof(VTYPE));

  hipMemcpy2D(d_synapse, pitch, synapse, Ni * sizeof(VTYPE), Ni * sizeof(VTYPE), Nn, hipMemcpyHostToDevice);
  hipMemcpy(d_neuron_i, neuron_i, Ni * sizeof(VTYPE), hipMemcpyHostToDevice);

  std::cout << "Simple version: \t";

/*
  for (int num_threads = 2; num_threads <= 1024; num_threads *= 2) {
    int num_blocks = Nn / num_threads;

    hipMemset(d_neuron_n, 0, Nn * sizeof(VTYPE));
    CUDA_timeit([&]() {
      GPU_classifier<<<num_blocks, num_threads>>>(d_synapse, d_neuron_i, d_neuron_n);
    });
    hipMemcpy(neuron_n3, d_neuron_n, Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);

    std::cout << "#threads = " << num_threads << ", #blocks = " << num_blocks << std::endl;
    compare(neuron_n, neuron_n3, Nn);
  }*/

  int num_threads = 8;
  int num_blocks = Nn / num_threads;

  hipMemset(d_neuron_n, 0, Nn * sizeof(VTYPE));
  CUDA_timeit([&]() {
    GPU_classifier<<<num_blocks, num_threads>>>(d_synapse, d_neuron_i, d_neuron_n, pitch);
  });
  hipMemcpy(neuron_n3, d_neuron_n, Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);

  compare(neuron_n, neuron_n3, Nn);
  std::cout << "Blocked version:\t";

  hipMemset(d_neuron_n, 0, Nn * sizeof(VTYPE));
  dim3 blockDim(1, Tn);
  dim3 gridDim(Ni/Ti, Nn/Tn);
  CUDA_timeit([&]() {
    GPU_classifier_tiled<<<gridDim, blockDim>>>(d_synapse, d_neuron_i, d_neuron_n, pitch);
  });
  auto err = hipGetLastError();
  if(err) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    return 0;
  }

  compare(neuron_n, neuron_n2, Nn);
}
