#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <functional>
#include "dnn.hpp"

using namespace std;

//Define the parameters if not defined externally
#ifndef Sy
  #define Sy 1
  #define Sx 1
#endif

#ifndef Tnn
  //Tiling Sizes
  #define Tnn 32
  #define Tn  16
  #define Ti  16
  
  #define Ty  8
  #define Tx  8
#endif

#define NYPAD (Ny+Ky)
#define NXPAD (Nx+Kx)

#define NYSCL (Ny/Sy)
#define NXSCL (Nx/Sx)

#define SYNAPSE_SIZE (1L*Ky*Kx*Nn*Ni)

VTYPE (*synapse)[Ky][Kx][Nn][Ni];
VTYPE (*neuron_i)[NYPAD][NXPAD][Ni];
VTYPE (*neuron_n)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n2)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n3)[NYSCL][NXSCL][Nn];

void convolution_tiled(const VTYPE synapse[Ky][Kx][Nn][Ni], 
                       const VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                       VTYPE neuron_n[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn] = {};

  for (int yy = 0; yy < Ny; yy += Ty) {
    for (int xx = 0; xx < Nx; xx += Tx) {
      for (int nnn = 0; nnn < Nn; nnn += Tnn) {
        int yout = yy / Sy;
        for (int y = yy; y < yy + Ty; y += Sy) { // tiling for y;
          int xout = xx / Sx;

          for (int x = xx; x < xx + Tx; x += Sx) { // tiling for x;

            for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
              memset(sum + nn, 0, Tn * sizeof(VTYPE));

              for (int ky = 0; ky < Ky; ky++) {  // sliding window;
                for (int kx = 0; kx < Kx; kx++) {

                  int ii = 0;
                  VTYPE sum_sc;

                  for (; ii < Ni -Ti+1; ii += Ti) {
                    for (int n = nn; n < nn + Tn; n++) {
                      sum_sc=0;
                      for (int i = ii; i < ii + Ti; i++) {
                        VTYPE sv = synapse[ky][kx][n][i];
                        VTYPE nv = neuron_i[ky + y][kx + x][i];
                        sum_sc += sv * nv;
                      }
                      sum[n] += sum_sc;
                    }
                  }
                }
              }

              //transfer
              for (int n = nn; n < nn + Tn; n++) {
                neuron_n[yout][xout][n] = transfer(sum[n]);
              }
            }
            xout++; 
          }
          yout++;
        }
      }
    }
  }
}

void convolution(const VTYPE synapse[Ky][Kx][Nn][Ni], 
                 const VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                 VTYPE neuron_n[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn] = {};

  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Ny; x += Sx) { // tiling for x;
      for (int nn = 0; nn < Nn; nn += Tn) {
        memset(sum + nn, 0, Tn * sizeof(VTYPE));

        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
            for (int n = nn; n < nn + Tn; n++)
              for (int i = 0; i < Ni; i++) {
                VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                sum[n] += sv * nv;
              }
        for (int n = nn; n < nn + Tn; n++) {
          neuron_n[yout][xout][n] = transfer(sum[n]);
        }
      }
      xout++; 
    }
    yout++;
  }
}


__global__ void GPU_convolution(const VTYPE synapse[Ky][Kx][Nn][Ni], 
                                const VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                                VTYPE neuron_n[NYSCL][NXSCL][Nn]/*,
                                size_t pitch*/) {
  const int t = blockIdx.z * blockDim.z + threadIdx.z;
  VTYPE sum = 0;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // sliding window;
  for (int ky = 0; ky < Ky; ky++) {
    for (int kx = 0; kx < Kx; kx++) {
      for (int i = 0; i < Ni; i++) {
        VTYPE sv = synapse[ky][kx][t][i];
        VTYPE nv = neuron_i[ky+y][kx+x][i];
        sum += sv * nv;
      }
    }
  }
  neuron_n[y][x][t] = GPU_transfer(sum);
}

int main(void) {
  std::cout << "------ Running CPU version ------" << std::endl;

  synapse   = (VTYPE (*)[Ky][Kx][Nn][Ni])   aligned_alloc(64, SYNAPSE_SIZE * sizeof(VTYPE));
  neuron_i  = (VTYPE (*)[NYPAD][NXPAD][Ni]) aligned_alloc(64, NYPAD * NXPAD * Ni * sizeof(VTYPE));
  neuron_n  = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));
  neuron_n3 = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));

  fill_random((VTYPE *) synapse, SYNAPSE_SIZE);
  fill_random((VTYPE *) neuron_i, NXPAD * NYPAD * Ni);

  std::cout << "Simple version: \t";
  auto f1 = std::bind(convolution, *synapse, *neuron_i, *neuron_n);
  timeit(f1);

  std::cout << "Tiled version:\t";  
  auto f2 = std::bind(convolution_tiled, *synapse, *neuron_i, *neuron_n2);
  timeit(f2);

  compare((VTYPE *)neuron_n, (VTYPE *)neuron_n2, NYSCL * NXSCL * Nn);

  std::cout << "------ Running GPU version ------" << std::endl;


/*
  // flatten synapse from 4D into 3D array
  hipExtent extent_synapse = make_hipExtent(Kx * sizeof(VTYPE), Ky, Nn * Ni);
  hipExtent extent_neuron_i = make_hipExtent(NXPAD * sizeof(VTYPE), NYPAD, Ni);
  hipExtent extent_neuron_n = make_hipExtent(NXSCL * sizeof(VTYPE), NYSCL, Ni);

  size_t pitch_synapse, pitch_neuron_i, pitch_neuron_n;
  hipPitchedPtr d_synapse, d_neuron_i, d_neuron_n;
  hipMalloc3D(&d_synapse, extent_synapse);
  hipMalloc3D(&d_neuron_i, extent_neuron_i);
  hipMalloc3D(&d_neuron_n, extent_neuron_n);

  hipMemcpy3DParms p_synapse = {}, p_neuron_i = {}, p_neruron_n = {};
  p_synapse.srcPtr = synapse;
  p_synapse.dstPtr = d_synapse;
  p_synapse.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(p_synapse);
*/
  /* use oridnary memory layout for the naive version now */
  VTYPE (*d_synapse) [Kx][Nn][Ni];
  VTYPE (*d_neuron_i) [NXPAD][Ni];
  VTYPE (*d_neuron_n) [NXSCL][Nn];
  hipMalloc(&d_synapse, Kx * Ky * Nn * Ni * sizeof(VTYPE));
  hipMalloc(&d_neuron_i, NXPAD * NYPAD * Ni * sizeof(VTYPE));
  hipMalloc(&d_neuron_n, NXSCL * NYSCL * Ni * sizeof(VTYPE));
  hipMemcpy(d_synapse, synapse, Kx * Ky * Nn * Ni * sizeof(VTYPE), hipMemcpyHostToDevice); 
  hipMemcpy(d_neuron_i, neuron_i, NYPAD * NXPAD * Ni * sizeof(VTYPE), hipMemcpyHostToDevice); 

  memset(neuron_n3, 0, NYSCL * NXSCL * Nn * sizeof(VTYPE));

  std::cout << "Simple version: \t";

//  for (int num_threads = 2; num_threads <= 1024; num_threads *= 2) {
//    int num_blocks = Nn / num_threads;
  
    hipMemset(d_neuron_n, 0, NYSCL * NXSCL * Nn * sizeof(VTYPE));
    CUDA_timeit([&]() {
      dim3 grid_size(NXSCL/Tx, NYSCL/Ty, Nn/Tn);
      dim3 block_size(Tx, Ty, Tn);
//      printf("%d %d %d %d %d %d\n", NXSCL/Tx, NYSCL/Ty, Nn/Tn, Tx, Ty, Tn);
      GPU_convolution<<<grid_size, block_size>>>(d_synapse, d_neuron_i, d_neuron_n/*, pitch*/);
    });
    hipMemcpy(neuron_n3, d_neuron_n, NYSCL * NXSCL * Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);
  auto err = hipGetLastError();
  if(err) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    return 0;
  } 
//    std::cout << "#threads = " << num_threads << ", #blocks = " << num_blocks << std::endl;
    compare((VTYPE *) neuron_n, (VTYPE *) neuron_n3, NYSCL * NXSCL * Nn);
//  }
  std::cout << "Tiled version:\t";  
}
