#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <functional>
#include "dnn.hpp"

using namespace std;

//Define the parameters if not defined externally
#ifndef Sy
  #define Sy 1
  #define Sx 1
#endif

#ifndef Tnn
  //Tiling Sizes
  #define Tnn 32
  #define Tn  16
  #define Ti  16
  
  #define Ty  8
  #define Tx  8
#endif

#define NYPAD (Ny+Ky)
#define NXPAD (Nx+Kx)

#define NYSCL (Ny/Sy)
#define NXSCL (Nx/Sx)

#define SYNAPSE_SIZE (1L*Ky*Kx*Nn*Ni)

VTYPE (*synapse)[Ky][Kx][Nn][Ni];
VTYPE (*neuron_i)[NYPAD][NXPAD][Ni];
VTYPE (*neuron_n)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n2)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n3)[NYSCL][NXSCL][Nn];

void convolution_layer_blocked(VTYPE synapse[Ky][Kx][Nn][Ni], 
                               VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                               VTYPE neuron_n[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn] = {};

  for (int yy = 0; yy < Ny; yy += Ty) {
    for (int xx = 0; xx < Nx; xx += Tx) {
      for (int nnn = 0; nnn < Nn; nnn += Tnn) {
        int yout = yy / Sy;
        for (int y = yy; y < yy + Ty; y += Sy) { // tiling for y;
          int xout = xx / Sx;

          for (int x = xx; x < xx + Tx; x += Sx) { // tiling for x;

            for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
              memset(sum + nn, 0, Tn * sizeof(VTYPE));

              for (int ky = 0; ky < Ky; ky++) {  // sliding window;
                for (int kx = 0; kx < Kx; kx++) {

                  int ii = 0;
                  VTYPE sum_sc;

                  for (; ii < Ni -Ti+1; ii += Ti) {
                    for (int n = nn; n < nn + Tn; n++) {
                      sum_sc=0;
                      for (int i = ii; i < ii + Ti; i++) {
                        VTYPE sv = synapse[ky][kx][n][i];
                        VTYPE nv = neuron_i[ky + y][kx + x][i];
                        sum_sc+=sv*nv;
                      }
                      sum[n]+=sum_sc;
                    }
                  }
                }
              }

              //transfer
              for (int n = nn; n < nn + Tn; n++) {
                neuron_n[yout][xout][n] = transfer(sum[n]);
              }
            }
            xout++; 
          }
          yout++;
        }
      }
    }
  }
}

void convolution_layer(VTYPE synapse[Ky][Kx][Nn][Ni], 
                       VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                       VTYPE neuron_n[NYSCL][NXSCL][Nn]) {
  VTYPE sum[Nn]={0};

  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Ny; x += Sx) { // tiling for x;
      for (int nn = 0; nn < Nn; nn += Tn) {
        memset(sum + nn, 0, Tn * sizeof(VTYPE));

        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
            for (int n = nn; n < nn + Tn; n++)
              for (int i = 0; i < Ni; i++) {
                VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                sum[n] += sv * nv;
              }
        for (int n = nn; n < nn + Tn; n++) {
          neuron_n[yout][xout][n] = transfer(sum[n]);
        }
      }
      xout++; 
    }
    yout++;
  }
}


__global__
void GPU_convolution_layer(const VTYPE synapse[Ky][Kx][Nn][Ni], 
                           const VTYPE neuron_i[NYPAD][NXPAD][Ni], 
                           VTYPE neuron_n[NYSCL][NXSCL][Nn],
                           size_t pitch) {
  // VTYPE sum[Nn]={0};
  const int t = blockIdx.x * blockDim.x + threadIdx.x;
  VTYPE *synapse_row = (VTYPE *)((char *)synapse + t * pitch);
  VTYPE *neuron_i_row = (VTYPE *)((char *)neuron_i + t * pitch);
  VTYPE *neuron_n_row = (VTYPE *)((char *)neuron_n + t * pitch);
  VTYPE sum = 0;
  int y = blockIdx.z / Ty + threadIdx.y;
  int x = blockIdx.z % Tx + threadIdx.x;

  // sliding window;
  for (int ky = 0; ky < Ky; ky++) {
    for (int kx = 0; kx < Kx; kx++) {
      for (int i = 0; i < Ni; i++) {
	VTYPE sv = synapse_row[blockIdx.x * SYNAPSE_SIZE + ky * (Kx * Nn * Ni) + kx * (Nn * Ni) + blockIdx.y * Ni + i];
	VTYPE nv = neuron_i_row[blockIdx.x * (NYPAD * NXPAD * Ni) + (ky + y) * NXPAD * Ni + (kx + x) * Ni + i];
	sum += sv*nv;
      }
    }
  }
  neuron_n_row[blockIdx.x * (NYSCL * NXSCL * Nn) + y * (NXSCL * Nn) + x * Nn + blockIdx.y]  = sum > 0 ? sum : sum/4; 
}

int main(void) {
  std::cout << "------ Running CPU version ------" << std::endl;

  synapse = (VTYPE (*)[Ky][Kx][Nn][Ni]) aligned_alloc(64, SYNAPSE_SIZE * sizeof(VTYPE));
  neuron_i = (VTYPE (*)[NYPAD][NXPAD][Ni]) aligned_alloc(64, NYPAD * NXPAD * Ni * sizeof(VTYPE));
  neuron_n = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));
  neuron_n2 = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));
  neuron_n3 = (VTYPE (*)[NYSCL][NXSCL][Nn]) aligned_alloc(64, NYSCL * NXSCL * Nn * sizeof(VTYPE));

  fill_random((VTYPE *) synapse, SYNAPSE_SIZE);
  fill_random((VTYPE *) neuron_i, NXPAD * NYPAD * Ni);

  std::cout << "Simple version: \t";
  auto f1 = std::bind(convolution_layer, *synapse, *neuron_i, *neuron_n);
  timeit(f1);

  std::cout << "Blocked version:\t";  
  auto f2 = std::bind(convolution_layer_blocked, *synapse, *neuron_i, *neuron_n2);
  timeit(f2);

  compare((VTYPE *) neuron_n, (VTYPE *) neuron_n2, NYSCL * NXSCL * Nn);

  std::cout << "------ Running GPU version ------" << std::endl;


  VTYPE (*d_synapse) [Kx][Nn][Ni];
  VTYPE (*d_neuron_i) [NXPAD][Ni];
  VTYPE (*d_neuron_n) [NXSCL][Nn];
  size_t pitch;

  hipMallocPitch((void **) &d_synapse, &pitch, sizeof(VTYPE) * Kx * Nn * Ni, Ky);
  hipMallocManaged((void **) &d_neuron_i, sizeof(VTYPE) * NYPAD * NXPAD * Ni);
  hipMallocManaged((void **) &d_neuron_n, sizeof(VTYPE) * NYSCL * NXSCL * Nn);

  hipMemcpy2D(d_synapse, pitch, synapse, Kx * sizeof(VTYPE), Nn * Ni  * sizeof(VTYPE), Ky,  hipMemcpyHostToDevice); 
  hipMemcpy(d_neuron_i, neuron_i, NYPAD * NXPAD * Ni * sizeof(VTYPE), hipMemcpyHostToDevice); 
  hipMemcpy(d_neuron_n, neuron_n, NYSCL * NXSCL * Nn * sizeof(VTYPE), hipMemcpyHostToDevice); 

  memset(neuron_n3, 0, NYSCL * NXSCL * Nn * sizeof(VTYPE));

  std::cout << "Simple version: \t";

  for (int num_threads = 2; num_threads <= 1024; num_threads *= 2) {
    int num_blocks = Nn / num_threads;
  
    hipMemset(d_neuron_n, 0, NYSCL * NXSCL * Nn * sizeof(VTYPE));
    CUDA_timeit([&]() {
      GPU_convolution_layer<<<num_blocks, num_threads>>>(d_synapse, d_neuron_i, d_neuron_n, pitch);
    });
    hipMemcpy(neuron_n3, d_neuron_n, NYSCL * NXSCL * Nn * sizeof(VTYPE), hipMemcpyDeviceToHost);
    std::cout << "#threads = " << num_threads << ", #blocks = " << num_blocks << std::endl;
    compare((VTYPE *) neuron_n, (VTYPE *) neuron_n3, NYSCL * NXSCL * Nn);
  }  
  std::cout << "Blocked version:\t";  
}



